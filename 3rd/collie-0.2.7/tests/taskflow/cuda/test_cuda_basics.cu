#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <collie/testing/doctest.h>
#include <collie/taskflow/taskflow.h>
#include <collie/taskflow/cuda/cudaflow.h>

// ----------------------------------------------------------------------------
// kernel helper
// ----------------------------------------------------------------------------
template <typename T>
__global__ void k_set(T* ptr, size_t N, T value) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < N) {
    ptr[i] = value;
  }
}

template <typename T>
__global__ void k_single_set(T* ptr, int i, T value) {
  ptr[i] = value;
}

template <typename T>
__global__ void k_add(T* ptr, size_t N, T value) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < N) {
    ptr[i] += value;
  }
}

template <typename T>
__global__ void k_single_add(T* ptr, int i, T value) {
  ptr[i] += value;
}

template <typename T>
void run_and_wait(T& cf) {
  collie::tf::cudaStream stream;
  cf.run(stream);
  stream.synchronize();
}

// --------------------------------------------------------
// Testcase: Empty
// --------------------------------------------------------

template <typename T>
void empty() {
  std::atomic<int> counter{0};
  
  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;

  taskflow.emplace([&](){ 
    T tf;
    ++counter; 
  });
  
  taskflow.emplace([&](){ 
    T tf;
    ++counter; 
  });
  
  taskflow.emplace([&](){ 
    T tf;
    ++counter; 
  });

  executor.run_n(taskflow, 100).wait();

  REQUIRE(counter == 300);
}

TEST_CASE("Empty" * doctest::timeout(300)) {
  empty<collie::tf::cudaFlow>();
}

TEST_CASE("EmptyCapture" * doctest::timeout(300)) {
  empty<collie::tf::cudaFlowCapturer>();
}

// ----------------------------------------------------------------------------
// Move Semantics
// ----------------------------------------------------------------------------

template <typename F>
void move_semantics() {

  unsigned N = 1024;
  
  F rhs;

  REQUIRE(rhs.num_tasks() == 0);
  REQUIRE(rhs.empty());
  REQUIRE(rhs.native_executable() == nullptr);

  // construct a cudaflow of three tasks
  auto cpu = static_cast<int*>(std::calloc(N, sizeof(int)));
  auto gpu = collie::tf::cuda_malloc_device<int>(N);
  dim3 g = {(N+255)/256, 1, 1};
  dim3 b = {256, 1, 1};
  auto h2d = rhs.copy(gpu, cpu, N);
  auto kernel = rhs.kernel(g, b, 0, k_add<int>, gpu, N, 17);
  auto d2h = rhs.copy(cpu, gpu, N);
  h2d.precede(kernel);
  kernel.precede(d2h);

  REQUIRE(rhs.num_tasks() == 3);
  REQUIRE(rhs.empty() == false);
  REQUIRE(rhs.native_executable() == nullptr);
  
  // construct a rhs
  F lhs( std::move(rhs) );

  REQUIRE(rhs.num_tasks() == 0);
  REQUIRE(rhs.empty());
  REQUIRE(rhs.native_executable() == nullptr);
  
  REQUIRE(lhs.num_tasks() == 3);
  REQUIRE(lhs.empty() == false);
  REQUIRE(lhs.native_executable() == nullptr);

  // assign lhs to rhs using move semantics
  rhs = std::move(lhs);
  
  REQUIRE(lhs.num_tasks() == 0);
  REQUIRE(lhs.empty());
  REQUIRE(lhs.native_executable() == nullptr);
  
  REQUIRE(rhs.num_tasks() == 3);
  REQUIRE(rhs.empty() == false);
  REQUIRE(rhs.native_executable() == nullptr);

  // run
  rhs.run(0);
  hipStreamSynchronize(0);

  auto native_graph = rhs.native_graph();
  auto native_executable = rhs.native_executable();

  REQUIRE(native_graph != nullptr);
  REQUIRE(native_executable != nullptr);
  REQUIRE(rhs.num_tasks() == 3);
  REQUIRE(rhs.empty() == false);
  REQUIRE(rhs.native_graph() != nullptr);
  REQUIRE(rhs.native_executable() != nullptr);
  REQUIRE(collie::tf::cuda_graph_get_num_nodes(rhs.native_graph()) == rhs.num_tasks());
  
  for(unsigned i=0; i<N; ++i) {
    REQUIRE(cpu[i] == 17);
  }

  // assign rhs to lhs using move semantics
  lhs = std::move(rhs);
  
  REQUIRE(lhs.num_tasks() == 3);
  REQUIRE(lhs.empty() == false);
  REQUIRE(lhs.native_graph() == native_graph);
  REQUIRE(lhs.native_executable() == native_executable);
  REQUIRE(collie::tf::cuda_graph_get_num_nodes(lhs.native_graph()) == lhs.num_tasks());
  
  REQUIRE(rhs.num_tasks() == 0);
  REQUIRE(rhs.empty());
  REQUIRE(rhs.native_graph() == nullptr);
  REQUIRE(rhs.native_executable() == nullptr);

  // run the flow again
  for(size_t j=2; j<=10; j++) {

    lhs.run(0);
    hipStreamSynchronize(0);
    
    for(unsigned i=0; i<N; ++i) {
      REQUIRE(cpu[i] == j*17);
    }
    
    REQUIRE(lhs.num_tasks() == 3);
    REQUIRE(lhs.empty() == false);
    REQUIRE(lhs.native_graph() == native_graph);
    REQUIRE(lhs.native_executable() == native_executable);
    REQUIRE(collie::tf::cuda_graph_get_num_nodes(lhs.native_graph()) == lhs.num_tasks());
    
    REQUIRE(rhs.num_tasks() == 0);
    REQUIRE(rhs.empty());
    REQUIRE(rhs.native_graph() == nullptr);
    REQUIRE(rhs.native_executable() == nullptr);
  }
}

TEST_CASE("cudaFlow.MoveSemantics" * doctest::timeout(300)) {
  move_semantics<collie::tf::cudaFlow>();
}

TEST_CASE("cudaFlowCapturer.MoveSemantics" * doctest::timeout(300)) {
  move_semantics<collie::tf::cudaFlowCapturer>();
}

// ----------------------------------------------------------------------------
// Standalone
// ----------------------------------------------------------------------------
template <typename T>
void standalone() {

  T cf;
  collie::tf::cudaStream stream;
  REQUIRE(cf.empty());

  unsigned N = 1024;
    
  auto cpu = static_cast<int*>(std::calloc(N, sizeof(int)));
  auto gpu = collie::tf::cuda_malloc_device<int>(N);

  dim3 g = {(N+255)/256, 1, 1};
  dim3 b = {256, 1, 1};
  auto h2d = cf.copy(gpu, cpu, N);
  auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, N, 17);
  auto d2h = cf.copy(cpu, gpu, N);
  h2d.precede(kernel);
  kernel.precede(d2h);
    
  for(unsigned i=0; i<N; ++i) {
    REQUIRE(cpu[i] == 0);
  }

  cf.run(stream);
  stream.synchronize();
  for(unsigned i=0; i<N; ++i) {
    REQUIRE(cpu[i] == 17);
  }
  
  for(size_t i=0; i<9; i++) {
    cf.run(stream);
  }
  stream.synchronize();

  for(unsigned i=0; i<N; ++i) {
    REQUIRE(cpu[i] == 170);
  }
  
  std::free(cpu);
  collie::tf::cuda_free(gpu);
}

TEST_CASE("Standalone.cudaFlow") {
  standalone<collie::tf::cudaFlow>();
}

TEST_CASE("Standalone.cudaCapturer") {
  standalone<collie::tf::cudaFlowCapturer>();
}



// --------------------------------------------------------
// Testcase: Set
// --------------------------------------------------------
template <typename T>
void set() {
    
  collie::tf::Executor executor;
  collie::tf::Taskflow taskflow;

  for(unsigned n=1; n<=123456; n = n*2 + 1) {

    taskflow.clear();
    
    T* cpu = nullptr;
    T* gpu = nullptr;

    auto cputask = taskflow.emplace([&](){
      cpu = static_cast<T*>(std::calloc(n, sizeof(T)));
      REQUIRE(hipMalloc(&gpu, n*sizeof(T)) == hipSuccess);
    });

    auto gputask = taskflow.emplace([&]() {
      collie::tf::cudaFlow cf;
      auto h2d = cf.copy(gpu, cpu, n);
      auto kernel = cf.kernel((n+255)/256, 256, 0, k_set<T>, gpu, n, (T)17);
      auto d2h = cf.copy(cpu, gpu, n);
      h2d.precede(kernel);
      kernel.precede(d2h);
      run_and_wait(cf);

      REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
    });

    cputask.precede(gputask);
    
    executor.run(taskflow).wait();

    for(unsigned i=0; i<n; ++i) {
      REQUIRE(cpu[i] == (T)17);
    }

    std::free(cpu);
    REQUIRE(hipFree(gpu) == hipSuccess);
  }
}

TEST_CASE("Set.i8" * doctest::timeout(300)) {
  set<int8_t>();
}

TEST_CASE("Set.i16" * doctest::timeout(300)) {
  set<int16_t>();
}

TEST_CASE("Set.i32" * doctest::timeout(300)) {
  set<int32_t>();
}

// --------------------------------------------------------
// Testcase: Add
// --------------------------------------------------------
template <typename T>
void add() {

  for(unsigned n=1; n<=123456; n = n*2 + 1) {
   
    collie::tf::Taskflow taskflow;
    collie::tf::Executor executor;
    
    T* cpu = nullptr;
    T* gpu = nullptr;
    
    auto cputask = taskflow.emplace([&](){
      cpu = static_cast<T*>(std::calloc(n, sizeof(T)));
      REQUIRE(hipMalloc(&gpu, n*sizeof(T)) == hipSuccess);
    });
    
    auto gputask = taskflow.emplace([&](){
      collie::tf::cudaFlow cf;
      dim3 g = {(n+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto h2d = cf.copy(gpu, cpu, n);
      auto ad1 = cf.kernel(g, b, 0, k_add<T>, gpu, n, 1);
      auto ad2 = cf.kernel(g, b, 0, k_add<T>, gpu, n, 2);
      auto ad3 = cf.kernel(g, b, 0, k_add<T>, gpu, n, 3);
      auto ad4 = cf.kernel(g, b, 0, k_add<T>, gpu, n, 4);
      auto d2h = cf.copy(cpu, gpu, n);
      h2d.precede(ad1);
      ad1.precede(ad2);
      ad2.precede(ad3);
      ad3.precede(ad4);
      ad4.precede(d2h);
      run_and_wait(cf);
      REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
    });

    cputask.precede(gputask);
    
    executor.run(taskflow).wait();

    for(unsigned i=0; i<n; ++i) {
      REQUIRE(cpu[i] == 10);
    }

    std::free(cpu);
    REQUIRE(hipFree(gpu) == hipSuccess);
  }
}

TEST_CASE("Add.i8" * doctest::timeout(300)) {
  add<int8_t>();
}

TEST_CASE("Add.i16" * doctest::timeout(300)) {
  add<int16_t>();
}

TEST_CASE("Add.i32" * doctest::timeout(300)) {
  add<int32_t>();
}

// TODO: 64-bit fail?
//TEST_CASE("Add.i64" * doctest::timeout(300)) {
//  add<int64_t>();
//}


// --------------------------------------------------------
// Testcase: Binary Set
// --------------------------------------------------------
template <typename T, typename F>
void bset() {

  const unsigned n = 10000;

  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;

  T* cpu = nullptr;
  T* gpu = nullptr;
  
  auto cputask = taskflow.emplace([&](){
    cpu = static_cast<T*>(std::calloc(n, sizeof(T)));
    REQUIRE(hipMalloc(&gpu, n*sizeof(T)) == hipSuccess);
  });

  auto gputask = taskflow.emplace([&]() {
    F cf;
    dim3 g = {1, 1, 1};
    dim3 b = {1, 1, 1};
    auto h2d = cf.copy(gpu, cpu, n);
    auto d2h = cf.copy(cpu, gpu, n);

    std::vector<collie::tf::cudaTask> tasks(n+1);

    for(unsigned i=1; i<=n; ++i) {
      tasks[i] = cf.kernel(g, b, 0, k_single_set<T>, gpu, i-1, (T)17);

      auto p = i/2;
      if(p != 0) {
        tasks[p].precede(tasks[i]);
      }

      tasks[i].precede(d2h);
      h2d.precede(tasks[i]);
    }

    run_and_wait(cf);
    REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
  });

  cputask.precede(gputask);
  
  executor.run(taskflow).wait();

  for(unsigned i=0; i<n; ++i) {
    REQUIRE(cpu[i] == (T)17);
  }

  std::free(cpu);
  REQUIRE(hipFree(gpu) == hipSuccess);
}

TEST_CASE("BSet.i8" * doctest::timeout(300)) {
  bset<int8_t, collie::tf::cudaFlow>();
}

TEST_CASE("BSet.i16" * doctest::timeout(300)) {
  bset<int16_t, collie::tf::cudaFlow>();
}

TEST_CASE("BSet.i32" * doctest::timeout(300)) {
  bset<int32_t, collie::tf::cudaFlow>();
}

TEST_CASE("CapturedBSet.i8" * doctest::timeout(300)) {
  bset<int8_t, collie::tf::cudaFlowCapturer>();
}

TEST_CASE("CapturedBSet.i16" * doctest::timeout(300)) {
  bset<int16_t, collie::tf::cudaFlowCapturer>();
}

TEST_CASE("CapturedBSet.i32" * doctest::timeout(300)) {
  bset<int32_t, collie::tf::cudaFlowCapturer>();
}

// --------------------------------------------------------
// Testcase: Memset
// --------------------------------------------------------

template <typename F>
void memset() {
  
  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;
  
  const int N = 100;

  int* cpu = new int [N];
  int* gpu = nullptr;
    
  REQUIRE(hipMalloc(&gpu, N*sizeof(int)) == hipSuccess);

  for(int r=1; r<=100; ++r) {

    int start = ::rand() % N;

    for(int i=0; i<N; ++i) {
      cpu[i] = 999;
    }
    
    taskflow.emplace([&](){
      F cf;
      dim3 g = {(unsigned)(N+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto kset = cf.kernel(g, b, 0, k_set<int>, gpu, N, 123);
      auto copy = cf.copy(cpu, gpu, N);
      auto zero = cf.memset(gpu+start, 0x3f, (N-start)*sizeof(int));
      kset.precede(zero);
      zero.precede(copy);
      run_and_wait(cf);
      REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
    });
    
    executor.run(taskflow).wait();

    for(int i=0; i<start; ++i) {
      REQUIRE(cpu[i] == 123);
    }
    for(int i=start; i<N; ++i) {
      REQUIRE(cpu[i] == 0x3f3f3f3f);
    }
  }
  
  delete [] cpu;
  REQUIRE(hipFree(gpu) == hipSuccess);
}

TEST_CASE("Memset" * doctest::timeout(300)) {
  memset<collie::tf::cudaFlow>();
}

TEST_CASE("CapturedMemset" * doctest::timeout(300)) {
  memset<collie::tf::cudaFlowCapturer>();
}

// --------------------------------------------------------
// Testcase: Memset0
// --------------------------------------------------------
template <typename T, typename F>
void memset0() {
  
  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;
  
  const int N = 97;

  T* cpu = new T [N];
  T* gpu = nullptr;
    
  REQUIRE(hipMalloc(&gpu, N*sizeof(T)) == hipSuccess);

  for(int r=1; r<=100; ++r) {

    int start = ::rand() % N;

    for(int i=0; i<N; ++i) {
      cpu[i] = (T)999;
    }
    
    taskflow.emplace([&](){
      F cf;
      dim3 g = {(unsigned)(N+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto kset = cf.kernel(g, b, 0, k_set<T>, gpu, N, (T)123);
      auto zero = cf.memset(gpu+start, (T)0, (N-start)*sizeof(T));
      auto copy = cf.copy(cpu, gpu, N);
      kset.precede(zero);
      zero.precede(copy);
      run_and_wait(cf);
      REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
    });
    
    executor.run(taskflow).wait();

    for(int i=0; i<start; ++i) {
      REQUIRE(std::fabs(cpu[i] - (T)123) < 1e-4);
    }
    for(int i=start; i<N; ++i) {
      REQUIRE(std::fabs(cpu[i] - (T)0) < 1e-4);
    }
  }
  
  delete [] cpu;
  REQUIRE(hipFree(gpu) == hipSuccess);
}

TEST_CASE("Memset0.i8") {
  memset0<int8_t, collie::tf::cudaFlow>();
}

TEST_CASE("Memset0.i16") {
  memset0<int16_t, collie::tf::cudaFlow>();
}

TEST_CASE("Memset0.i32") {
  memset0<int32_t, collie::tf::cudaFlow>();
}

TEST_CASE("Memset0.f32") {
  memset0<float, collie::tf::cudaFlow>();
}

TEST_CASE("Memset0.f64") {
  memset0<double, collie::tf::cudaFlow>();
}

TEST_CASE("CapturedMemset0.i8") {
  memset0<int8_t, collie::tf::cudaFlowCapturer>();
}

TEST_CASE("CapturedMemset0.i16") {
  memset0<int16_t, collie::tf::cudaFlowCapturer>();
}

TEST_CASE("CapturedMemset0.i32") {
  memset0<int32_t, collie::tf::cudaFlowCapturer>();
}

TEST_CASE("CapturedMemset0.f32") {
  memset0<float, collie::tf::cudaFlowCapturer>();
}

TEST_CASE("CapturedMemset0.f64") {
  memset0<double, collie::tf::cudaFlowCapturer>();
}

// --------------------------------------------------------
// Testcase: Memcpy
// --------------------------------------------------------
template <typename T, typename F>
void memcpy() {
  
  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;
  
  const int N = 97;

  T* cpu = new T [N];
  T* gpu = nullptr;
    
  REQUIRE(hipMalloc(&gpu, N*sizeof(T)) == hipSuccess);

  for(int r=1; r<=100; ++r) {

    int start = ::rand() % N;

    for(int i=0; i<N; ++i) {
      cpu[i] = (T)999;
    }
    
    taskflow.emplace([&](){
      F cf;
      dim3 g = {(unsigned)(N+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto kset = cf.kernel(g, b, 0, k_set<T>, gpu, N, (T)123);
      auto zero = cf.memset(gpu+start, (T)0, (N-start)*sizeof(T));
      auto copy = cf.memcpy(cpu, gpu, N*sizeof(T));
      kset.precede(zero);
      zero.precede(copy);
      run_and_wait(cf);
      REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
    });
    
    executor.run(taskflow).wait();

    for(int i=0; i<start; ++i) {
      REQUIRE(std::fabs(cpu[i] - (T)123) < 1e-4);
    }
    for(int i=start; i<N; ++i) {
      REQUIRE(std::fabs(cpu[i] - (T)0) < 1e-4);
    }
  }
  
  delete [] cpu;
  REQUIRE(hipFree(gpu) == hipSuccess);
}

TEST_CASE("Memcpy.i8") {
  memcpy<int8_t, collie::tf::cudaFlow>();
}

TEST_CASE("Memcpy.i16") {
  memcpy<int16_t, collie::tf::cudaFlow>();
}

TEST_CASE("Memcpy.i32") {
  memcpy<int32_t, collie::tf::cudaFlow>();
}

TEST_CASE("Memcpy.f32") {
  memcpy<float, collie::tf::cudaFlow>();
}

TEST_CASE("Memcpy.f64") {
  memcpy<double, collie::tf::cudaFlow>();
}

TEST_CASE("CapturedMemcpy.i8") {
  memcpy<int8_t, collie::tf::cudaFlowCapturer>();
}

TEST_CASE("CapturedMemcpy.i16") {
  memcpy<int16_t, collie::tf::cudaFlowCapturer>();
}

TEST_CASE("CapturedMemcpy.i32") {
  memcpy<int32_t, collie::tf::cudaFlowCapturer>();
}

TEST_CASE("CapturedMemcpy.f32") {
  memcpy<float, collie::tf::cudaFlowCapturer>();
}

TEST_CASE("CapturedMemcpy.f64") {
  memcpy<double, collie::tf::cudaFlowCapturer>();
}

// --------------------------------------------------------
// Testcase: fill
// --------------------------------------------------------
template <typename T>
void fill(T value) {
  
  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;
  
  const int N = 107;

  T* cpu = new T [N];
  T* gpu = nullptr;
    
  REQUIRE(hipMalloc(&gpu, N*sizeof(T)) == hipSuccess);

  for(int r=1; r<=100; ++r) {

    int start = ::rand() % N;

    for(int i=0; i<N; ++i) {
      cpu[i] = (T)999;
    }
    
    taskflow.emplace([&](){

      collie::tf::cudaFlow cf;

      dim3 g = {(unsigned)(N+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto kset = cf.kernel(g, b, 0, k_set<T>, gpu, N, (T)123);
      auto fill = cf.fill(gpu+start, value, (N-start));
      auto copy = cf.copy(cpu, gpu, N);
      kset.precede(fill);
      fill.precede(copy);

      run_and_wait(cf);
      REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
    });
    
    executor.run(taskflow).wait();

    for(int i=0; i<start; ++i) {
      REQUIRE(std::fabs(cpu[i] - (T)123) < 1e-4);
    }
    for(int i=start; i<N; ++i) {
      REQUIRE(std::fabs(cpu[i] - value) < 1e-4);
    }
  }

  delete [] cpu;
  REQUIRE(hipFree(gpu) == hipSuccess);
}

TEST_CASE("Fill.i8") {
  fill<int8_t>(+123);
  fill<int8_t>(-123);
}

TEST_CASE("Fill.i16") {
  fill<int16_t>(+12345);
  fill<int16_t>(-12345);
}

TEST_CASE("Fill.i32") {
  fill<int32_t>(+123456789);
  fill<int32_t>(-123456789);
}

TEST_CASE("Fill.f32") {
  fill<float>(+123456789.0f);
  fill<float>(-123456789.0f);
}

// --------------------------------------------------------
// Testcase: Zero
// --------------------------------------------------------
template <typename T>
void zero() {
  
  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;
  
  const int N = 100;

  T* cpu = new T [N];
  T* gpu = nullptr;
    
  REQUIRE(hipMalloc(&gpu, N*sizeof(T)) == hipSuccess);

  for(int r=1; r<=100; ++r) {

    int start = ::rand() % N;

    for(int i=0; i<N; ++i) {
      cpu[i] = (T)999;
    }
    
    taskflow.emplace([&](){

      collie::tf::cudaFlow cf;

      dim3 g = {(unsigned)(N+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto kset = cf.kernel(g, b, 0, k_set<T>, gpu, N, (T)123);
      auto zero = cf.zero(gpu+start, (N-start));
      auto copy = cf.copy(cpu, gpu, N);
      kset.precede(zero);
      zero.precede(copy);

      run_and_wait(cf);
      REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
    });
    
    executor.run(taskflow).wait();

    for(int i=0; i<start; ++i) {
      REQUIRE(std::fabs(cpu[i] - (T)123) < 1e-4);
    }
    for(int i=start; i<N; ++i) {
      REQUIRE(std::fabs(cpu[i] - (T)0) < 1e-4);
    }
  }

  delete [] cpu;
  REQUIRE(hipFree(gpu) == hipSuccess);
}

TEST_CASE("Zero.i8") {
  zero<int8_t>();
}

TEST_CASE("Zero.i16") {
  zero<int16_t>();
}

TEST_CASE("Zero.i32") {
  zero<int32_t>();
}

TEST_CASE("Zero.f32") {
  zero<float>();
}

// --------------------------------------------------------
// Testcase: Barrier
// --------------------------------------------------------
template <typename T>
void barrier() {

  const unsigned n = 1000;
  
  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;
  
  T* cpu = nullptr;
  T* gpu = nullptr;

  auto cputask = taskflow.emplace([&](){
    cpu = static_cast<T*>(std::calloc(n, sizeof(T)));
    REQUIRE(hipMalloc(&gpu, n*sizeof(T)) == hipSuccess);
  });

  auto gputask = taskflow.emplace([&]() {
    
    collie::tf::cudaFlow cf;

    dim3 g = {1, 1, 1};
    dim3 b = {1, 1, 1};
    auto br1 = cf.noop();
    auto br2 = cf.noop();
    auto br3 = cf.noop();
    auto h2d = cf.copy(gpu, cpu, n);
    auto d2h = cf.copy(cpu, gpu, n);

    h2d.precede(br1);

    for(unsigned i=0; i<n; ++i) {
      auto k1 = cf.kernel(g, b, 0, k_single_set<T>, gpu, i, (T)17);
      k1.succeed(br1)
        .precede(br2);

      auto k2 = cf.kernel(g, b, 0, k_single_add<T>, gpu, i, (T)3);
      k2.succeed(br2)
        .precede(br3);
    }

    br3.precede(d2h);

    run_and_wait(cf);
    REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
  });

  cputask.precede(gputask);
  
  executor.run(taskflow).wait();

  for(unsigned i=0; i<n; ++i) {
    REQUIRE(cpu[i] == (T)20);
  }

  std::free(cpu);
  REQUIRE(hipFree(gpu) == hipSuccess);
}

TEST_CASE("Barrier.i8" * doctest::timeout(300)) {
  barrier<int8_t>();
}

TEST_CASE("Barrier.i16" * doctest::timeout(300)) {
  barrier<int16_t>();
}

TEST_CASE("Barrier.i32" * doctest::timeout(300)) {
  barrier<int32_t>();
}

// ----------------------------------------------------------------------------
// NestedRuns
// ----------------------------------------------------------------------------
  
template <typename F>
void nested_runs() {

  int* cpu = nullptr;
  int* gpu = nullptr;

  constexpr unsigned n = 1000;

  cpu = static_cast<int*>(std::calloc(n, sizeof(int)));
  REQUIRE(hipMalloc(&gpu, n*sizeof(int)) == hipSuccess);

  struct A {

    collie::tf::Executor executor;
    collie::tf::Taskflow taskflow;

    void run(int* cpu, int* gpu, unsigned n) {
      taskflow.clear();

      auto A1 = taskflow.emplace([&]() {  
        F cf;
        cf.copy(gpu, cpu, n);
        run_and_wait(cf);
        REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
      });

      auto A2 = taskflow.emplace([&]() { 
        F cf;
        dim3 g = {(n+255)/256, 1, 1};
        dim3 b = {256, 1, 1};
        cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
        run_and_wait(cf);
        REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
      });

      auto A3 = taskflow.emplace([&] () {
        F cf;
        cf.copy(cpu, gpu, n);
        run_and_wait(cf);
        REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
      });

      A1.precede(A2);
      A2.precede(A3);

      executor.run_n(taskflow, 10).wait();
    }

  };
  
  struct B {

    collie::tf::Taskflow taskflow;
    collie::tf::Executor executor;

    A a;

    void run(int* cpu, int* gpu, unsigned n) {

      taskflow.clear();
      
      auto B0 = taskflow.emplace([] () {});
      auto B1 = taskflow.emplace([&] () { 
        F cf;
        dim3 g = {(n+255)/256, 1, 1};
        dim3 b = {256, 1, 1};
        auto h2d = cf.copy(gpu, cpu, n);
        auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
        auto d2h = cf.copy(cpu, gpu, n);
        h2d.precede(kernel);
        kernel.precede(d2h);
        run_and_wait(cf);
        REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
      });
      auto B2 = taskflow.emplace([&] () { a.run(cpu, gpu, n); });
      auto B3 = taskflow.emplace([&] () { 
        for(unsigned i=0; i<n; ++i) {
          cpu[i]++;
        }
      });
      
      B0.precede(B1);
      B1.precede(B2);
      B2.precede(B3);

      executor.run_n(taskflow, 100).wait();
    }
  };

  B b;
  b.run(cpu, gpu, n);

  for(unsigned i=0; i<n; i++) {
    REQUIRE(cpu[i] == 1200);
  }
    
  REQUIRE(hipFree(gpu) == hipSuccess);
  std::free(cpu);
}

TEST_CASE("NestedRuns" * doctest::timeout(300)) {
  nested_runs<collie::tf::cudaFlow>();
}

TEST_CASE("CapturedNestedRuns" * doctest::timeout(300)) {
  nested_runs<collie::tf::cudaFlowCapturer>();
}

/*
// ----------------------------------------------------------------------------
// WorkerID
// ----------------------------------------------------------------------------

void worker_id(unsigned N, unsigned M) {
  
  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor(N + M);

  REQUIRE(executor.num_workers() == (N + M));

  const unsigned s = 100;

  for(unsigned k=0; k<s; ++k) {
    
    auto cputask = taskflow.emplace([&](){
      auto id = executor.this_worker_id();
      REQUIRE(id >= 0);
      REQUIRE(id <  N+M);
    });
    
    auto gputask = taskflow.emplace([&](collie::tf::cudaFlow&) {
      auto id = executor.this_worker_id();
      REQUIRE(id >= 0);
      REQUIRE(id <  N+M);
    });

    auto chktask = taskflow.emplace([&] () {
      auto id = executor.this_worker_id();
      REQUIRE(id >= 0);
      REQUIRE(id <  N+M);
    });
    
    taskflow.emplace([&](collie::tf::cudaFlow&) {
      auto id = executor.this_worker_id();
      REQUIRE(id >= 0);
      REQUIRE(id <  N+M);
    });
    
    taskflow.emplace([&]() {
      auto id = executor.this_worker_id();
      REQUIRE(id >= 0);
      REQUIRE(id <  N+M);
    });

    auto subflow = taskflow.emplace([&](collie::tf::Subflow& sf){
      auto id = executor.this_worker_id();
      REQUIRE(id >= 0);
      REQUIRE(id <  N+M);
      auto t1 = sf.emplace([&](){
        auto id = executor.this_worker_id();
        REQUIRE(id >= 0);
        REQUIRE(id <  N+M);
      });
      auto t2 = sf.emplace([&](collie::tf::cudaFlow&){
        auto id = executor.this_worker_id();
        REQUIRE(id >= 0);
        REQUIRE(id <  N+M);
      });
      t1.precede(t2);
    });

    cputask.precede(gputask);
    gputask.precede(chktask);
    chktask.precede(subflow);
  }

  executor.run_n(taskflow, 10).wait();
}

TEST_CASE("WorkerID.1C1G") {
  worker_id(1, 1);
}

TEST_CASE("WorkerID.1C2G") {
  worker_id(1, 2);
}

TEST_CASE("WorkerID.1C3G") {
  worker_id(1, 3);
}

TEST_CASE("WorkerID.1C4G") {
  worker_id(1, 4);
}

TEST_CASE("WorkerID.2C1G") {
  worker_id(2, 1);
}

TEST_CASE("WorkerID.2C2G") {
  worker_id(2, 2);
}

TEST_CASE("WorkerID.2C3G") {
  worker_id(2, 3);
}

TEST_CASE("WorkerID.2C4G") {
  worker_id(2, 4);
}

TEST_CASE("WorkerID.3C1G") {
  worker_id(3, 1);
}

TEST_CASE("WorkerID.3C2G") {
  worker_id(3, 2);
}

TEST_CASE("WorkerID.3C3G") {
  worker_id(3, 3);
}

TEST_CASE("WorkerID.3C4G") {
  worker_id(3, 4);
}

TEST_CASE("WorkerID.4C1G") {
  worker_id(4, 1);
}

TEST_CASE("WorkerID.4C2G") {
  worker_id(4, 2);
}

TEST_CASE("WorkerID.4C3G") {
  worker_id(4, 3);
}

TEST_CASE("WorkerID.4C4G") {
  worker_id(4, 4);
} */

// ----------------------------------------------------------------------------
// Multiruns
// ----------------------------------------------------------------------------

void multiruns(unsigned N, unsigned M) {

  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor(N + M);

  const unsigned n = 1000;
  const unsigned s = 100;

  int *cpu[s] = {0};
  int *gpu[s] = {0};

  for(unsigned k=0; k<s; ++k) {
    
    int number = ::rand()%100;

    auto cputask = taskflow.emplace([&, k](){
      cpu[k] = static_cast<int*>(std::calloc(n, sizeof(int)));
      REQUIRE(hipMalloc(&gpu[k], n*sizeof(int)) == hipSuccess);
    });
    
    auto gputask = taskflow.emplace([&, k, number]() {
      collie::tf::cudaFlow cf;
      dim3 g = {(n+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto h2d = cf.copy(gpu[k], cpu[k], n);
      auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu[k], n, number);
      auto d2h = cf.copy(cpu[k], gpu[k], n);
      h2d.precede(kernel);
      kernel.precede(d2h);
      run_and_wait(cf);
      REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
    });

    auto chktask = taskflow.emplace([&, k, number] () {
      for(unsigned i=0; i<n; ++i) {
        REQUIRE(cpu[k][i] == number);
      }
    });

    cputask.precede(gputask);
    gputask.precede(chktask);

  }

  executor.run(taskflow).wait();
}

TEST_CASE("Multiruns.1C1G") {
  multiruns(1, 1);
}

TEST_CASE("Multiruns.1C2G") {
  multiruns(1, 2);
}

TEST_CASE("Multiruns.1C3G") {
  multiruns(1, 3);
}

TEST_CASE("Multiruns.1C4G") {
  multiruns(1, 4);
}

TEST_CASE("Multiruns.2C1G") {
  multiruns(2, 1);
}

TEST_CASE("Multiruns.2C2G") {
  multiruns(2, 2);
}

TEST_CASE("Multiruns.2C3G") {
  multiruns(2, 3);
}

TEST_CASE("Multiruns.2C4G") {
  multiruns(2, 4);
}

TEST_CASE("Multiruns.3C1G") {
  multiruns(3, 1);
}

TEST_CASE("Multiruns.3C2G") {
  multiruns(3, 2);
}

TEST_CASE("Multiruns.3C3G") {
  multiruns(3, 3);
}

TEST_CASE("Multiruns.3C4G") {
  multiruns(3, 4);
}

TEST_CASE("Multiruns.4C1G") {
  multiruns(4, 1);
}

TEST_CASE("Multiruns.4C2G") {
  multiruns(4, 2);
}

TEST_CASE("Multiruns.4C3G") {
  multiruns(4, 3);
}

TEST_CASE("Multiruns.4C4G") {
  multiruns(4, 4);
}

// ----------------------------------------------------------------------------
// Subflow
// ----------------------------------------------------------------------------

template <typename F>
void subflow() {
  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;
    
  int* cpu = nullptr;
  int* gpu = nullptr;
  
  const unsigned n = 1000;

  auto partask = taskflow.emplace([&](collie::tf::Subflow& sf){

    auto cputask = sf.emplace([&](){
      cpu = static_cast<int*>(std::calloc(n, sizeof(int)));
      REQUIRE(hipMalloc(&gpu, n*sizeof(int)) == hipSuccess);
    });
    
    auto gputask = sf.emplace([&]() {
      F cf;
      dim3 g = {(n+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto h2d = cf.copy(gpu, cpu, n);
      auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
      auto d2h = cf.copy(cpu, gpu, n);
      h2d.precede(kernel);
      kernel.precede(d2h);
      run_and_wait(cf);
      REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
    });

    cputask.precede(gputask);
  });
    
  auto chktask = taskflow.emplace([&](){
    for(unsigned i=0; i<n ;++i){
      REQUIRE(cpu[i] == 1);
    }
    REQUIRE(hipFree(gpu) == hipSuccess);
    std::free(cpu);
  });

  partask.precede(chktask);

  executor.run(taskflow).wait();

}

TEST_CASE("Subflow" * doctest::timeout(300)) {
  subflow<collie::tf::cudaFlow>();
}

TEST_CASE("CapturedSubflow" * doctest::timeout(300)) {
  subflow<collie::tf::cudaFlowCapturer>();
}

// ----------------------------------------------------------------------------
// NestedSubflow
// ----------------------------------------------------------------------------

template <typename F>
void nested_subflow() {

  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;
    
  int* cpu = nullptr;
  int* gpu = nullptr;
  
  const unsigned n = 1000;
    
  auto cputask = taskflow.emplace([&](){
    cpu = static_cast<int*>(std::calloc(n, sizeof(int)));
    REQUIRE(hipMalloc(&gpu, n*sizeof(int)) == hipSuccess);
  });

  auto partask = taskflow.emplace([&](collie::tf::Subflow& sf){
    
    auto gputask1 = sf.emplace([&]() {
      F cf;
      dim3 g = {(n+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto h2d = cf.copy(gpu, cpu, n);
      auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
      auto d2h = cf.copy(cpu, gpu, n);
      h2d.precede(kernel);
      kernel.precede(d2h);
      run_and_wait(cf);
      REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
    });

    auto subtask1 = sf.emplace([&](collie::tf::Subflow& sf2) {
      auto gputask2 = sf2.emplace([&]() {
        F cf;
        dim3 g = {(n+255)/256, 1, 1};
        dim3 b = {256, 1, 1};
        auto h2d = cf.copy(gpu, cpu, n);
        auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
        auto d2h = cf.copy(cpu, gpu, n);
        h2d.precede(kernel);
        kernel.precede(d2h);
        run_and_wait(cf);
        REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
      });
      
      auto subtask2 = sf2.emplace([&](collie::tf::Subflow& sf3){
        sf3.emplace([&]() {
          F cf;
          dim3 g = {(n+255)/256, 1, 1};
          dim3 b = {256, 1, 1};
          auto h2d = cf.copy(gpu, cpu, n);
          auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
          auto d2h = cf.copy(cpu, gpu, n);
          h2d.precede(kernel);
          kernel.precede(d2h);
          run_and_wait(cf);
          REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
        });
      });

      gputask2.precede(subtask2);
    });

    gputask1.precede(subtask1);
  });
    
  auto chktask = taskflow.emplace([&](){
    for(unsigned i=0; i<n ;++i){
      REQUIRE(cpu[i] == 3);
    }
    REQUIRE(hipFree(gpu) == hipSuccess);
    std::free(cpu);
  });

  partask.precede(chktask)
         .succeed(cputask);

  executor.run(taskflow).wait();

}

TEST_CASE("NestedSubflow" * doctest::timeout(300) ) {
  nested_subflow<collie::tf::cudaFlow>();
}

TEST_CASE("CapturedNestedSubflow" * doctest::timeout(300) ) {
  nested_subflow<collie::tf::cudaFlowCapturer>();
}


// ----------------------------------------------------------------------------
// DetachedSubflow
// ----------------------------------------------------------------------------

template <typename F>
void detached_subflow() {

  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;
    
  int* cpu = nullptr;
  int* gpu = nullptr;
  
  const unsigned n = 1000;

  taskflow.emplace([&](collie::tf::Subflow& sf){

    auto cputask = sf.emplace([&](){
      cpu = static_cast<int*>(std::calloc(n, sizeof(int)));
      REQUIRE(hipMalloc(&gpu, n*sizeof(int)) == hipSuccess);
    });
    
    auto gputask = sf.emplace([&]() {
      F cf;
      dim3 g = {(n+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto h2d = cf.copy(gpu, cpu, n);
      auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
      auto d2h = cf.copy(cpu, gpu, n);
      h2d.precede(kernel);
      kernel.precede(d2h);
      run_and_wait(cf);
      REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
    });

    cputask.precede(gputask);

    sf.detach();
  });
    
  executor.run(taskflow).wait();
  
  for(unsigned i=0; i<n ;++i){
    REQUIRE(cpu[i] == 1);
  }
  REQUIRE(hipFree(gpu) == hipSuccess);
  std::free(cpu);
}

TEST_CASE("DetachedSubflow" * doctest::timeout(300)) {
  detached_subflow<collie::tf::cudaFlow>();
}

TEST_CASE("CapturedDetachedSubflow" * doctest::timeout(300)) {
  detached_subflow<collie::tf::cudaFlowCapturer>();
}

// ----------------------------------------------------------------------------
// Conditional GPU tasking
// ----------------------------------------------------------------------------

template <typename F>
void loop() {

  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;

  const unsigned n = 1000;
    
  int* cpu = nullptr;
  int* gpu = nullptr;

  auto cputask = taskflow.emplace([&](){
    cpu = static_cast<int*>(std::calloc(n, sizeof(int)));
    REQUIRE(hipMalloc(&gpu, n*sizeof(int)) == hipSuccess);
  });

  auto gputask = taskflow.emplace([&]() {
    F cf;
    dim3 g = {(n+255)/256, 1, 1};
    dim3 b = {256, 1, 1};
    auto h2d = cf.copy(gpu, cpu, n);
    auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
    auto d2h = cf.copy(cpu, gpu, n);
    h2d.precede(kernel);
    kernel.precede(d2h);
    run_and_wait(cf);
    REQUIRE(cf.num_tasks() == collie::tf::cuda_graph_get_num_nodes(cf.native_graph()));
  });

  auto condition = taskflow.emplace([&cpu, round=0] () mutable {
    ++round;
    for(unsigned i=0; i<n; ++i) {
      REQUIRE(cpu[i] == round);
    }
    return round >= 100;
  });

  auto freetask = taskflow.emplace([&](){
    REQUIRE(hipFree(gpu) == hipSuccess);
    std::free(cpu);
  });

  cputask.precede(gputask);
  gputask.precede(condition);
  condition.precede(gputask, freetask);
  
  executor.run(taskflow).wait();
}

TEST_CASE("Loop" * doctest::timeout(300)) {
  loop<collie::tf::cudaFlow>();
}

TEST_CASE("CapturedLoop" * doctest::timeout(300)) {
  loop<collie::tf::cudaFlowCapturer>();
}


// ----------------------------------------------------------------------------
// Predicate
// ----------------------------------------------------------------------------

TEST_CASE("Predicate") {

  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;

  const unsigned n = 1000;
    
  int* cpu = nullptr;
  int* gpu = nullptr;

  auto cputask = taskflow.emplace([&](){
    cpu = static_cast<int*>(std::calloc(n, sizeof(int)));
    REQUIRE(hipMalloc(&gpu, n*sizeof(int)) == hipSuccess);
    REQUIRE(hipMemcpy(gpu, cpu, n*sizeof(int), hipMemcpyHostToDevice) == hipSuccess);
  });

  auto gputask = taskflow.emplace([&]() {
    collie::tf::cudaFlow cf;
    dim3 g = {(n+255)/256, 1, 1};
    dim3 b = {256, 1, 1};
    auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
    auto copy = cf.copy(cpu, gpu, n);
    kernel.precede(copy);

    collie::tf::cudaStream stream;
    for(int i=0; i<100; i++) {
      cf.run(stream);
    }
    stream.synchronize();
  });

  auto freetask = taskflow.emplace([&](){
    for(unsigned i=0; i<n; ++i) {
      REQUIRE(cpu[i] == 100);
    }
    REQUIRE(hipFree(gpu) == hipSuccess);
    std::free(cpu);
  });

  cputask.precede(gputask);
  gputask.precede(freetask);
  
  executor.run(taskflow).wait();
}

// ----------------------------------------------------------------------------
// Repeat
// ----------------------------------------------------------------------------

TEST_CASE("Repeat") {

  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;

  const unsigned n = 1000;
    
  int* cpu = nullptr;
  int* gpu = nullptr;

  auto cputask = taskflow.emplace([&](){
    cpu = static_cast<int*>(std::calloc(n, sizeof(int)));
    REQUIRE(hipMalloc(&gpu, n*sizeof(int)) == hipSuccess);
    REQUIRE(hipMemcpy(gpu, cpu, n*sizeof(int), hipMemcpyHostToDevice) == hipSuccess);
  });

  auto gputask = taskflow.emplace([&]() {
    collie::tf::cudaFlow cf;
    dim3 g = {(n+255)/256, 1, 1};
    dim3 b = {256, 1, 1};
    auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
    auto copy = cf.copy(cpu, gpu, n);
    kernel.precede(copy);
    
    collie::tf::cudaStream stream;
    for(int i=0; i<100; i++) {
      cf.run(stream);
    }
    stream.synchronize();
  });

  auto freetask = taskflow.emplace([&](){
    for(unsigned i=0; i<n; ++i) {
      REQUIRE(cpu[i] == 100);
    }
    REQUIRE(hipFree(gpu) == hipSuccess);
    std::free(cpu);
  });

  cputask.precede(gputask);
  gputask.precede(freetask);
  
  executor.run(taskflow).wait();
}


