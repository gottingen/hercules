#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <collie/testing/doctest.h>
#include <collie/taskflow/taskflow.h>
#include <collie/taskflow/cuda/cudaflow.h>
#include <collie/taskflow/cuda/algorithm/for_each.hpp>

constexpr float eps = 0.0001f;

template <typename T>
void run_and_wait(T& cf) {
  collie::tf::cudaStream stream;
  cf.run(stream);
  stream.synchronize();
}

// ----------------------------------------------------------------------------
// for_each_index
// ----------------------------------------------------------------------------

template <typename T>
void cuda_for_each_index() {

  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;
  
  for(int n=0; n<=1234567; n = (n<=100) ? n+1 : n*2 + 1) {

    taskflow.emplace([n](){
      collie::tf::cudaStream stream;
      collie::tf::cudaDefaultExecutionPolicy policy(stream);

      auto g_data = collie::tf::cuda_malloc_shared<T>(n);
      for(int i=0; i<n; i++) {
        g_data[i] = 0;
      }

      collie::tf::cuda_for_each_index(policy,
        0, n, 1, [g_data] __device__ (int i) { g_data[i] = 12222; }
      );

      stream.synchronize();

      for(int i=0; i<n; i++) {
        REQUIRE(std::fabs(g_data[i] - (T)12222) < eps);
      }

      collie::tf::cuda_free(g_data);
    });
  }

  executor.run(taskflow).wait();
}

TEST_CASE("cuda_for_each_index.int" * doctest::timeout(300)) {
  cuda_for_each_index<int>();
}

TEST_CASE("cuda_for_each_index.float" * doctest::timeout(300)) {
  cuda_for_each_index<float>();
}

TEST_CASE("cuda_for_each_index.double" * doctest::timeout(300)) {
  cuda_for_each_index<double>();
}

// ----------------------------------------------------------------------------
// for_each_index
// ----------------------------------------------------------------------------

template <typename T, typename F>
void cudaflow_for_each_index() {
    
  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;

  for(int n=1; n<=1234567; n = (n<=100) ? n+1 : n*2 + 1) {
    
    taskflow.emplace([n](){

      auto cpu = static_cast<T*>(std::calloc(n, sizeof(T)));
      
      T* gpu = nullptr;
      REQUIRE(hipMalloc(&gpu, n*sizeof(T)) == hipSuccess);

      F cf;
      auto d2h = cf.copy(cpu, gpu, n);
      auto h2d = cf.copy(gpu, cpu, n);
      auto kernel = cf.for_each_index(
        0, n, 1, [gpu] __device__ (int i) { gpu[i] = 65536; }
      );
      h2d.precede(kernel);
      d2h.succeed(kernel);

      run_and_wait(cf);

      for(int i=0; i<n; i++) {
        REQUIRE(std::fabs(cpu[i] - (T)65536) < eps);
      }
      
      // update
      cf.for_each_index(kernel,
        0, n, 1, [gpu] __device__ (int i) { gpu[i] = (T)100; }
      );

      run_and_wait(cf);
      
      for(int j=0; j<n; j++) {
        REQUIRE(std::fabs(cpu[j] - (T)100) < eps);
      }

      std::free(cpu);
      REQUIRE(hipFree(gpu) == hipSuccess); 
    });
  }

  executor.run(taskflow).wait();
}

TEST_CASE("cudaFlow.for_each_index.int" * doctest::timeout(300)) {
  cudaflow_for_each_index<int, collie::tf::cudaFlow>();
}

TEST_CASE("cudaFlow.for_each_index.float" * doctest::timeout(300)) {
  cudaflow_for_each_index<float, collie::tf::cudaFlow>();
}

TEST_CASE("cudaFlow.for_each_index.double" * doctest::timeout(300)) {
  cudaflow_for_each_index<double, collie::tf::cudaFlow>();
}

TEST_CASE("cudaFlowCapturer.for_each_index.int" * doctest::timeout(300)) {
  cudaflow_for_each_index<int, collie::tf::cudaFlowCapturer>();
}

TEST_CASE("cudaFlowCapturer.for_each_index.float" * doctest::timeout(300)) {
  cudaflow_for_each_index<float, collie::tf::cudaFlowCapturer>();
}

TEST_CASE("cudaFlowCapturer.for_each_index.double" * doctest::timeout(300)) {
  cudaflow_for_each_index<double, collie::tf::cudaFlowCapturer>();
}

