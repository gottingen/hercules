#include "hip/hip_runtime.h"
// This program performs a simple single-precision Ax+Y operation
// using cudaFlow and verifies its result.

#include <collie/taskflow/taskflow.h>
#include <collie/taskflow/cuda/cudaflow.h>

// Kernel: saxpy
__global__ void saxpy(int n, float a, float *x, float *y) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = a*x[i] + y[i];
  }
}

// Function: main
int main() {
  
  const unsigned N = 1<<20;

  collie::tf::Taskflow taskflow ("saxpy-flow");
  collie::tf::Executor executor;

  std::vector<float> hx, hy;

  float* dx {nullptr};
  float* dy {nullptr};
  
  // allocate x
  auto allocate_x = taskflow.emplace([&]() {
    std::cout << "allocating host x and device x ...\n";
    hx.resize(N, 1.0f);
    hipMalloc(&dx, N*sizeof(float));
  }).name("allocate_x");

  // allocate y
  auto allocate_y = taskflow.emplace([&]() {
    std::cout << "allocating host y and device y ...\n";
    hy.resize(N, 2.0f);
    hipMalloc(&dy, N*sizeof(float));
  }).name("allocate_y");
  
  // saxpy cudaFlow
  auto cudaflow = taskflow.emplace([&]() {
    
    std::cout << "running cudaflow ...\n";

    collie::tf::cudaFlow cf;
    auto h2d_x = cf.copy(dx, hx.data(), N).name("h2d_x");
    auto h2d_y = cf.copy(dy, hy.data(), N).name("h2d_y");
    auto d2h_x = cf.copy(hx.data(), dx, N).name("d2h_x");
    auto d2h_y = cf.copy(hy.data(), dy, N).name("d2h_y");
    auto kernel = cf.kernel((N+255)/256, 256, 0, saxpy, N, 2.0f, dx, dy)
                    .name("saxpy");
    kernel.succeed(h2d_x, h2d_y)
          .precede(d2h_x, d2h_y);
    
    std::cout << "launching cudaflow ...\n";
    collie::tf::cudaStream stream;
    cf.run(stream);
    stream.synchronize();
    
    // visualize this cudaflow
    cf.dump(std::cout);

  }).name("saxpy");

  cudaflow.succeed(allocate_x, allocate_y);

  // Add a verification task
  auto verifier = taskflow.emplace([&](){
    float max_error = 0.0f;
    for (size_t i = 0; i < N; i++) {
      max_error = std::max(max_error, abs(hx[i]-1.0f));
      max_error = std::max(max_error, abs(hy[i]-4.0f));
    }
    std::cout << "saxpy finished with max error: " << max_error << '\n';
  }).succeed(cudaflow).name("verify");

  // free memory
  auto deallocate_x = taskflow.emplace([&](){
    std::cout << "deallocating device x ...\n";
    hipFree(dx);
  }).name("deallocate_x");
  
  auto deallocate_y = taskflow.emplace([&](){
    std::cout << "deallocating device y ...\n";
    hipFree(dy);
  }).name("deallocate_y");

  verifier.precede(deallocate_x, deallocate_y);

  executor.run(taskflow).wait();

  std::cout << "dumping the taskflow ...\n";
  taskflow.dump(std::cout);

  return 0;
}

