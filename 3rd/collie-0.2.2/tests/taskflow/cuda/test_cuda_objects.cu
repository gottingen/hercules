#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <collie/testing/doctest.h>
#include <collie/taskflow/taskflow.h>
#include <collie/taskflow/cuda/cudaflow.h>

TEST_CASE("cuda.version" * doctest::timeout(300) ) {
  REQUIRE(collie::tf::cuda_get_driver_version() > 0);
  REQUIRE(collie::tf::cuda_get_runtime_version() > 0);
}

TEST_CASE("cuda.device" * doctest::timeout(300) ) {

  REQUIRE(collie::tf::cuda_get_num_devices() > 0);
  REQUIRE(collie::tf::cuda_get_device() >= 0);

  size_t num_devices = tf::cuda_get_num_devices();

  for(size_t c=0; c<num_devices; c++) {
    tf::cuda_set_device(c);
    REQUIRE(collie::tf::cuda_get_device() == c);
    
    for(size_t d=0; d<num_devices; d++) {
      REQUIRE(collie::tf::cuda_get_device_max_threads_per_block(d) > 0);
      REQUIRE(collie::tf::cuda_get_device_max_x_dim_per_block(d) > 0);
      REQUIRE(collie::tf::cuda_get_device_max_y_dim_per_block(d) > 0);
      REQUIRE(collie::tf::cuda_get_device_max_z_dim_per_block(d) > 0);
      REQUIRE(collie::tf::cuda_get_device_max_x_dim_per_grid(d) > 0);
      REQUIRE(collie::tf::cuda_get_device_max_y_dim_per_grid(d) > 0);
      REQUIRE(collie::tf::cuda_get_device_max_z_dim_per_grid(d) > 0);
      REQUIRE(collie::tf::cuda_get_device_warp_size(d) > 0);
      REQUIRE(collie::tf::cuda_get_device_max_shm_per_block(d) > 0);
      REQUIRE(collie::tf::cuda_get_device_compute_capability_major(d) > 0);
      REQUIRE(collie::tf::cuda_get_device_compute_capability_minor(d) >= 0);
      REQUIRE_NOTHROW(collie::tf::cuda_get_device_unified_addressing(d));
    }
  }
  
  // going back to device 0
  tf::cuda_set_device(0);
}

// ----------------------------------------------------------------------------
// stream
// ----------------------------------------------------------------------------

TEST_CASE("cudaStream" * doctest::timeout(300)) {
  
  // create a new stream s1 inside
  tf::cudaStream s1;
  
  // create another stream s2 from the outside
  hipStream_t s2_source;
  hipStreamCreate(&s2_source);
  tf::cudaStream s2(s2_source);
  
  REQUIRE(s2 == s2_source);

  hipStream_t s1_source = s1;
  REQUIRE(s1 == s1_source);

  // query status
  REQUIRE(hipStreamQuery(s1) == hipSuccess);
  REQUIRE(hipStreamQuery(s2) == hipSuccess);

  s1 = std::move(s2);

  REQUIRE(s2 == nullptr);
  REQUIRE(s1 == s2_source);
  REQUIRE(hipStreamQuery(s1) == hipSuccess);

}

// ----------------------------------------------------------------------------
// event
// ----------------------------------------------------------------------------

TEST_CASE("cudaEvent" * doctest::timeout(300)) {
  
  // create a new event e1 inside
  tf::cudaEvent e1;
  
  // create another event e2 from the outside
  hipEvent_t e2_source;
  hipEventCreate(&e2_source);
  tf::cudaEvent e2(e2_source);
  
  REQUIRE(e2 == e2_source);

  hipEvent_t e1_source = e1;
  REQUIRE(e1 == e1_source);

  // query status
  REQUIRE(hipEventQuery(e1) == hipSuccess);
  REQUIRE(hipEventQuery(e2) == hipSuccess);

  e1 = std::move(e2);

  REQUIRE(e2 == nullptr);
  REQUIRE(e1 == e2_source);
  REQUIRE(hipEventQuery(e1) == hipSuccess);
  REQUIRE(hipEventQuery(e2) != hipSuccess);
}

// ----------------------------------------------------------------------------
// CUDA Graph
// ----------------------------------------------------------------------------

TEST_CASE("cudaGraph" * doctest::timeout(300)) {
  
  // create a new graph g1 inside
  tf::cudaGraph g1;
  
  hipGraph_t g1_source = g1;
  REQUIRE(g1 == g1_source);
  
  // create another graph g2 from the outside
  hipGraph_t g2_source;
  hipGraphCreate(&g2_source, 0);
  tf::cudaGraph g2(g2_source);
  
  REQUIRE(g2 == g2_source);

  g1 = std::move(g2);

  REQUIRE(g2 == nullptr);
  REQUIRE(g1 == g2_source);

  // reassign g1 (now holding g2_source) to g2
  g2.reset(g1.release());
  REQUIRE(g1 == nullptr);
  REQUIRE(g2 == g2_source);

  // clear
  g2.clear();
  g1.clear();

  REQUIRE(g1 == nullptr);
  REQUIRE(g2 == nullptr);
}


