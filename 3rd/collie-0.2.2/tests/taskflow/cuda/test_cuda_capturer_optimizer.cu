#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <collie/testing/doctest.h>
#include <collie/taskflow/taskflow.h>
#include <collie/taskflow/cuda/cudaflow.h>
#include <collie/taskflow/cuda/algorithm/for_each.hpp>

#include "./details/graph_executor.hpp"
#include "./details/tree.hpp"
#include "./details/random_DAG.hpp"
#include "./details/tree.hpp"
#include "./details/diamond.hpp"

// ----------------------------------------------------------------------------
// Graph traversal
// ----------------------------------------------------------------------------
template <typename GRAPH, typename OPT, typename... OPT_Args>
void traversal(OPT_Args&&... args) {
  for(int i = 0; i < 13; ++i) {
    Graph* g;
    if constexpr(std::is_same_v<GRAPH, Tree>) {
      g = new Tree(::rand() % 3 + 1, ::rand() % 5 + 1);
    }
    else if constexpr(std::is_same_v<GRAPH, RandomDAG>) {
      g = new RandomDAG(::rand() % 10 + 1, ::rand() % 10 + 1, ::rand() % 10 + 1);
    }
    else if constexpr(std::is_same_v<GRAPH, Diamond>) {
      g = new Diamond(::rand() % 10 + 1, ::rand() % 10 + 1);
    }
    GraphExecutor<OPT> executor(*g, 0); 
    executor.traversal(std::forward<OPT_Args>(args)...);

    REQUIRE(g->traversed());
    delete g;
  }

}

TEST_CASE("cudaFlowCapturer.tree.Sequential") {
  traversal<Tree, collie::tf::cudaFlowSequentialOptimizer>();
}

TEST_CASE("cudaFlowCapturer.tree.RoundRobin.1") {
  traversal<Tree, collie::tf::cudaFlowRoundRobinOptimizer>(1);
}

TEST_CASE("cudaFlowCapturer.tree.RoundRobin.2") {
  traversal<Tree, collie::tf::cudaFlowRoundRobinOptimizer>(2);
}

TEST_CASE("cudaFlowCapturer.tree.RoundRobin.3") {
  traversal<Tree, collie::tf::cudaFlowRoundRobinOptimizer>(3);
}

TEST_CASE("cudaFlowCapturer.tree.RoundRobin.4") {
  traversal<Tree, collie::tf::cudaFlowRoundRobinOptimizer>(4);
}

//TEST_CASE("cudaFlowCapturer.tree.Greedy.1") {
//  traversal<Tree, collie::tf::cudaGreedyCapturing>(1);
//}
//
//TEST_CASE("cudaFlowCapturer.tree.Greedy.2") {
//  traversal<Tree, collie::tf::cudaGreedyCapturing>(2);
//}
//
//TEST_CASE("cudaFlowCapturer.tree.Greedy.3") {
//  traversal<Tree, collie::tf::cudaGreedyCapturing>(3);
//}
//
//TEST_CASE("cudaFlowCapturer.tree.Greedy.4") {
//  traversal<RandomDAG, collie::tf::cudaGreedyCapturing>(4);
//}

TEST_CASE("cudaFlowCapturer.randomDAG.Sequential") {
  traversal<RandomDAG,collie::tf::cudaFlowSequentialOptimizer>();
}

TEST_CASE("cudaFlowCapturer.randomDAG.RoundRobin.1") {
  traversal<RandomDAG, collie::tf::cudaFlowRoundRobinOptimizer>(1);
}

TEST_CASE("cudaFlowCapturer.randomDAG.RoundRobin.2") {
  traversal<RandomDAG, collie::tf::cudaFlowRoundRobinOptimizer>(2);
}

TEST_CASE("cudaFlowCapturer.randomDAG.RoundRobin.3") {
  traversal<RandomDAG, collie::tf::cudaFlowRoundRobinOptimizer>(3);
}

TEST_CASE("cudaFlowCapturer.randomDAG.RoundRobin.4") {
  traversal<RandomDAG, collie::tf::cudaFlowRoundRobinOptimizer>(4);
}

//TEST_CASE("cudaFlowCapturer.randomDAG.Greedy.1") {
//  traversal<RandomDAG, collie::tf::cudaGreedyCapturing>(1);
//}
//
//TEST_CASE("cudaFlowCapturer.randomDAG.Greedy.2") {
//  traversal<RandomDAG, collie::tf::cudaGreedyCapturing>(2);
//}
//
//TEST_CASE("cudaFlowCapturer.randomDAG.Greedy.3") {
//  traversal<RandomDAG, collie::tf::cudaGreedyCapturing>(3);
//}
//
//TEST_CASE("cudaFlowCapturer.randomDAG.Greedy.4") {
//  traversal<RandomDAG, collie::tf::cudaGreedyCapturing>(4);
//}

TEST_CASE("cudaFlowCapturer.diamond.Sequential") {
  traversal<Diamond, collie::tf::cudaFlowSequentialOptimizer>();
}

TEST_CASE("cudaFlowCapturer.diamond.RoundRobin.1") {
  traversal<Diamond, collie::tf::cudaFlowRoundRobinOptimizer>(1);
}

TEST_CASE("cudaFlowCapturer.diamond.RoundRobin.2") {
  traversal<Diamond, collie::tf::cudaFlowRoundRobinOptimizer>(2);
}

TEST_CASE("cudaFlowCapturer.diamond.RoundRobin.3") {
  traversal<Diamond, collie::tf::cudaFlowRoundRobinOptimizer>(3);
}

TEST_CASE("cudaFlowCapturer.diamond.RoundRobin.4") {
  traversal<Diamond, collie::tf::cudaFlowRoundRobinOptimizer>(4);
}

//TEST_CASE("cudaFlowCapturer.diamond.Greedy.1") {
//  traversal<Diamond, collie::tf::cudaGreedyCapturing>(1);
//}
//
//TEST_CASE("cudaFlowCapturer.diamond.Greedy.2") {
//  traversal<Diamond, collie::tf::cudaGreedyCapturing>(2);
//}
//
//TEST_CASE("cudaFlowCapturer.diamond.Greedy.3") {
//  traversal<Diamond, collie::tf::cudaGreedyCapturing>(3);
//}
//
//TEST_CASE("cudaFlowCapturer.diamond.Greedy.4") {
//  traversal<Diamond, collie::tf::cudaGreedyCapturing>(4);
//}

//------------------------------------------------------
// dependencies
//------------------------------------------------------

template <typename OPT, typename... OPT_Args>
void dependencies(OPT_Args ...args) {
  
  for(int t = 0; t < 17; ++t) {
    int num_partitions = ::rand() % 5 + 1;
    int num_iterations = ::rand() % 7 + 1;

    Diamond g(num_partitions, num_iterations);

    collie::tf::cudaFlowCapturer cf;
    cf.make_optimizer<OPT>(std::forward<OPT_Args>(args)...);

    int* inputs{nullptr};
    REQUIRE(hipMallocManaged(&inputs, num_partitions * sizeof(int)) == hipSuccess);
    REQUIRE(hipMemset(inputs, 0, num_partitions * sizeof(int)) == hipSuccess);

    std::vector<std::vector<collie::tf::cudaTask>> tasks;
    tasks.resize(g.get_size());

    for(size_t l = 0; l < g.get_size(); ++l) {
      tasks[l].resize((g.get_graph())[l].size());
      for(size_t i = 0; i < (g.get_graph())[l].size(); ++i) {
        
        if(l % 2 == 1) {
          tasks[l][i] = cf.single_task([inputs, i] __device__ () {
            inputs[i]++;
          });
        }
        else {
          tasks[l][i] = cf.on([=](hipStream_t stream){
            cuda_for_each(
              collie::tf::cudaDefaultExecutionPolicy(stream), inputs, inputs + num_partitions, 
              [] __device__ (int& v) { v*=2; }
            );
          });
        }
      }
    }

    for(size_t l = 0; l < g.get_size() - 1; ++l) {
      for(size_t i = 0; i < (g.get_graph())[l].size(); ++i) {
        for(auto&& out_node: g.at(l, i).out_nodes) {
          tasks[l][i].precede(tasks[l + 1][out_node]);
        }
      }
    }

    collie::tf::cudaStream stream;
    cf.run(stream);
    stream.synchronize();
    
    int result = 2;
    for(int i = 1; i < num_iterations; ++i) {
      result = result * 2 + 2;
    }

    for(int i = 0; i < num_partitions; ++i) {
      REQUIRE(inputs[i] == result);
    }

    REQUIRE(hipFree(inputs) == hipSuccess);
  }
}

TEST_CASE("cudaFlowCapturer.dependencies.diamond.Sequential") {
  dependencies<collie::tf::cudaFlowSequentialOptimizer>();
}

TEST_CASE("cudaFlowCapturer.dependencies.diamond.RoundRobin.1") {
  dependencies<collie::tf::cudaFlowRoundRobinOptimizer>(1);
}

TEST_CASE("cudaFlowCapturer.dependencies.diamond.RoundRobin.2") {
  dependencies<collie::tf::cudaFlowRoundRobinOptimizer>(2);
}

TEST_CASE("cudaFlowCapturer.dependencies.diamond.RoundRobin.3") {
  dependencies<collie::tf::cudaFlowRoundRobinOptimizer>(3);
}

TEST_CASE("cudaFlowCapturer.dependencies.diamond.RoundRobin.4") {
  dependencies<collie::tf::cudaFlowRoundRobinOptimizer>(4);
}

//TEST_CASE("cudaFlowCapturer.dependencies.diamond.Greedy.1") {
//  dependencies<collie::tf::cudaGreedyCapturing>(1);
//}
//
//TEST_CASE("cudaFlowCapturer.dependencies.diamond.Greedy.2") {
//  dependencies<collie::tf::cudaGreedyCapturing>(2);
//}
//
//TEST_CASE("cudaFlowCapturer.dependencies.diamond.Greedy.3") {
//  dependencies<collie::tf::cudaGreedyCapturing>(3);
//}
//
//TEST_CASE("cudaFlowCapturer.dependencies.diamond.Greedy.4") {
//  dependencies<collie::tf::cudaGreedyCapturing>(4);
//}
