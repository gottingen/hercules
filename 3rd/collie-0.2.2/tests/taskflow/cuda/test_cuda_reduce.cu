#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <collie/testing/doctest.h>
#include <collie/taskflow/taskflow.h>
#include <collie/taskflow/cuda/cudaflow.h>
#include <collie/taskflow/cuda/algorithm/reduce.hpp>

// ----------------------------------------------------------------------------
// cuda_reduce_bufsz
// ----------------------------------------------------------------------------

TEST_CASE("cuda_reduce.BufferSize") {

  using P = collie::tf::cudaExecutionPolicy<32, 3>;
  
  // within one block
  for(unsigned i=0; i<=P::nv; i++) {
    REQUIRE(P::reduce_bufsz<int>(i) == 0);
  }

  // two blocks
  for(unsigned i=P::nv+1; i<=2*P::nv; i++) {
    REQUIRE(P::reduce_bufsz<int>(i) == 2*sizeof(int));
  }
  
  // three blocks
  for(unsigned i=2*P::nv+1; i<=3*P::nv; i++) {
    REQUIRE(P::reduce_bufsz<int>(i) == 3*sizeof(int));
  }

  REQUIRE(
    P::reduce_bufsz<int>(P::nv*P::nv) == P::nv*sizeof(int)
  );

  REQUIRE(
    P::reduce_bufsz<int>(P::nv*P::nv+1) == (P::nv + 3)*sizeof(int)
  );

  REQUIRE(
    P::reduce_bufsz<int>(P::nv*P::nv*2) == (2*P::nv + 2)*sizeof(int)
  );
  
}

// ----------------------------------------------------------------------------
// cuda_reduce
// ----------------------------------------------------------------------------

template <typename T>
void cuda_reduce() {

  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;
  
  for(int n=0; n<=1234567; n = (n<=100) ? n+1 : n*2 + 1) {
    taskflow.emplace([n](){
      collie::tf::cudaStream stream;
      collie::tf::cudaDefaultExecutionPolicy policy(stream);
      
      unsigned bufsz = policy.reduce_bufsz<T>(n);

      T gold {1000};

      auto gpu = collie::tf::cuda_malloc_shared<T>(n);
      auto res = collie::tf::cuda_malloc_shared<T>(1);
      auto buf = collie::tf::cuda_malloc_shared<T>(bufsz);
      for(int i=0; i<n; i++) {
        gpu[i] = i;
        gold += i;
      }
      *res = T{1000};  // initial value
      
      // reduce
      collie::tf::cuda_reduce(policy,
        gpu, gpu + n, res, [] __device__ (T a, T b) { return a + b; }, buf
      );
      stream.synchronize();

      REQUIRE(*res == gold);
      
      // uninitialized reduce
      collie::tf::cuda_uninitialized_reduce(policy,
        gpu, gpu + n, res, [] __device__ (T a, T b) { return a + b; }, buf
      );
      stream.synchronize();
      
      if(n == 0) {
        REQUIRE(*res == 1000);
      }
      else {
        REQUIRE(*res == gold - 1000);
      }

      REQUIRE(hipFree(gpu) == hipSuccess);
      REQUIRE(hipFree(res) == hipSuccess);
      REQUIRE(hipFree(buf) == hipSuccess);
    });
  }

  executor.run(taskflow).wait();
}

TEST_CASE("cuda_reduce.int" * doctest::timeout(300)) {
  cuda_reduce<int>();
}

// ----------------------------------------------------------------------------
// cuda_transform_reduce
// ----------------------------------------------------------------------------

template <typename T>
void cuda_transform_reduce() {

  collie::tf::Taskflow taskflow;
  collie::tf::Executor executor;
  
  for(int n=0; n<=1234567; n = (n<=100) ? n+1 : n*2 + 1) {
    taskflow.emplace([n](){
      collie::tf::cudaStream stream;
      collie::tf::cudaDefaultExecutionPolicy policy(stream);
      
      unsigned bufsz = policy.reduce_bufsz<T>(n);

      T gold {1000};

      auto gpu = collie::tf::cuda_malloc_shared<T>(n);
      auto res = collie::tf::cuda_malloc_shared<T>(1);
      auto buf = collie::tf::cuda_malloc_shared<T>(bufsz);
      for(int i=0; i<n; i++) {
        gpu[i] = i;
        gold += (-i);
      }
      *res = T{1000};  // initial value
      
      // reduce
      collie::tf::cuda_transform_reduce(policy,
        gpu, gpu + n, res, 
        [] __device__ (T a, T b) { return a + b; }, 
        [] __device__ (T a)      { return -a; }, 
        buf
      );
      stream.synchronize();

      REQUIRE(*res == gold);
      
      // uninitialized reduce
      collie::tf::cuda_uninitialized_transform_reduce(policy,
        gpu, gpu + n, res, 
        [] __device__ (T a, T b) { return a + b; }, 
        [] __device__ (T a)      { return -a; },
        buf
      );
      stream.synchronize();
      
      if(n == 0) {
        REQUIRE(*res == 1000);
      }
      else {
        REQUIRE(*res == gold - 1000);
      }

      REQUIRE(hipFree(gpu) == hipSuccess);
      REQUIRE(hipFree(res) == hipSuccess);
      REQUIRE(hipFree(buf) == hipSuccess);
    });
  }

  executor.run(taskflow).wait();
}

TEST_CASE("cuda_transform_reduce.int" * doctest::timeout(300)) {
  cuda_transform_reduce<int>();
}







