#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <collie/testing/doctest.h>
#include <collie/taskflow/taskflow.h>
#include <collie/taskflow/cuda/cudaflow.h>

void __global__ testKernel() {}

TEST_CASE("cudaFlowCapturer.noEventError") {
  collie::tf::cudaFlow f;
  f.capture([](collie::tf::cudaFlowCapturer& cpt) {
    cpt.on([] (hipStream_t stream) {
      testKernel<<<256,256,0,stream>>>();
    });
    REQUIRE((hipGetLastError() == hipSuccess));
  });
  REQUIRE((hipGetLastError() == hipSuccess));
}
